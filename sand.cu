#include "hip/hip_runtime.h"
///nvcc -o fil main.cu -O3 -m=64 -arch=compute_61 -code=sm_61 -Xptxas -allow-expensive-optimizations=true -Xptxas -v
#include <iostream>
#include <chrono>
#include <fstream>
#include <algorithm>
#include <inttypes.h>
#include <bitset>
#include <iostream>
#include <vector>
#include <map>
#include <iomanip>
#include <fstream>
#include <chrono>
#include <mutex>
#include <time.h>
#include "lcg.h"

#ifdef BOINC
  #include "boinc_api.h"
#if defined _WIN32 || defined _WIN64
  #include "boinc_win.h"
#endif
#endif

uint64_t millis() {return (std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch())).count();}


#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
    exit(code);
  }
}



// ===== LCG IMPLEMENTATION ===== //

namespace java_lcg { //region Java LCG
    #define Random uint64_t
    #define RANDOM_MULTIPLIER 0x5DEECE66DULL
    #define RANDOM_ADDEND 0xBULL
    #define RANDOM_MASK ((1ULL << 48u) - 1)
    #define get_random(seed) ((Random)((seed ^ RANDOM_MULTIPLIER) & RANDOM_MASK))


    __host__ __device__ __forceinline__ static int32_t random_next(Random *random, int bits) {
        *random = (*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
        return (int32_t) (*random >> (48u - bits));
    }
    __device__ __forceinline__ static int32_t random_next_int(Random *random, const uint16_t bound) {
        int32_t r = random_next(random, 31);
        const uint16_t m = bound - 1u;
        if ((bound & m) == 0) {
            r = (int32_t) ((bound * (uint64_t) r) >> 31u);
        } else {
            for (int32_t u = r;
                 u - (r = u % bound) + m < 0;
                 u = random_next(random, 31));
        }
        return r;
    }
    
    __device__ __host__ __forceinline__ static int32_t random_next_int_nonpow(Random *random, const uint16_t bound) {
        int32_t r = random_next(random, 31);
        const uint16_t m = bound - 1u;
        for (int32_t u = r;
             u - (r = u % bound) + m < 0;
             u = random_next(random, 31));
      return r;
    }
    __host__ __device__ __forceinline__ static double next_double(Random *random) {
        return (double) ((((uint64_t) ((uint32_t) random_next(random, 26)) << 27u)) + random_next(random, 27)) / (double)(1ULL << 53);
    }
    __host__ __device__ __forceinline__ static uint64_t random_next_long (Random *random) {
        return (((uint64_t)random_next(random, 32)) << 32u) + (int32_t)random_next(random, 32);
    }
    __host__ __device__ __forceinline__ static void advance2(Random *random) {
        *random = (*random * 0xBB20B4600A69LLU + 0x40942DE6BALLU) & RANDOM_MASK;
    }
    __host__ __device__ __forceinline__ static void advance3759(Random *random) {
        *random = (*random * 0x6FE85C031F25LLU + 0x8F50ECFF899LLU) & RANDOM_MASK;
    }

}
using namespace java_lcg;


namespace device_intrinsics { //region DEVICE INTRINSICS
    #define DEVICE_STATIC_INTRINSIC_QUALIFIERS  static __device__ __forceinline__

    #if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
    #define PXL_GLOBAL_PTR   "l"
    #else
    #define PXL_GLOBAL_PTR   "r"
    #endif

    DEVICE_STATIC_INTRINSIC_QUALIFIERS void __prefetch_local_l1(const void* const ptr)
    {
      asm("prefetch.local.L1 [%0];" : : PXL_GLOBAL_PTR(ptr));
    }

    DEVICE_STATIC_INTRINSIC_QUALIFIERS void __prefetch_global_uniform(const void* const ptr)
    {
      asm("prefetchu.L1 [%0];" : : PXL_GLOBAL_PTR(ptr));
    }

    DEVICE_STATIC_INTRINSIC_QUALIFIERS void __prefetch_local_l2(const void* const ptr)
    {
      asm("prefetch.local.L2 [%0];" : : PXL_GLOBAL_PTR(ptr));
    }

    #if __CUDA__ < 10
    #define __ldg(ptr) (*(ptr))
    #endif
}
using namespace device_intrinsics;






#define BLOCK_SIZE (128)
//#define BLOCK_SIZE (128)
#define WORK_SIZE_BITS 16
#define SEEDS_PER_CALL ((1ULL << (WORK_SIZE_BITS)) * (BLOCK_SIZE))
//#define SEEDS_PER_CALL 8000000




//Specifying where the (1 = dirt/grass, 0 = sand) is

// This will match the seed 76261196830436 (not pack.png ofc)
// Double match: 76261206560653 (almost 100% confirmed, sans very last bit of sand in first match)
// Triple match: 76273693341674 (100% match)
__constant__ int CHUNK_X = 6;
__constant__ int CHUNK_X_2 = 6;
__constant__ int CHUNK_X_3 = 5;
#define CHUNK_Z -1
#define CHUNK_Z_2 -2
#define CHUNK_Z_3 -1

#define INNER_X_START 4
#define INNER_Z_START 0

#define INNER_X_END 13
#define INNER_Z_END 2
__constant__ uint8_t DIRT_HEIGHT_2D[INNER_Z_END - INNER_Z_START + 1][INNER_X_END - INNER_X_START + 1] = {{1,15,15,15,1,15,0,15,15,15},
                                                                                                         {15,1,15,15,15,1,15,1,15,15},
                                                                                                         {15,15,1,1,15,15,1,1,1,0}};
__constant__ double LocalNoise2D[INNER_Z_END - INNER_Z_START + 1][INNER_X_END - INNER_X_START + 1];

#define EARLY_RETURN (INNER_Z_END * 16 + INNER_X_END)




#define INNER_X_START_2 0
#define INNER_Z_START_2 6

#define INNER_X_END_2 9
#define INNER_Z_END_2 15

__constant__ uint8_t DIRT_HEIGHT_2D_2[INNER_Z_END_2 - INNER_Z_START_2 + 1][INNER_X_END_2 - INNER_X_START_2 + 1] = {{0,15,15,15,15,15,15,15,15,15},
                                                                                                                   {15,0,0,15,15,15,15,15,15,15},
                                                                                                                   {0,15,15,0,15,15,15,15,15,15},
                                                                                                                   {15,1,15,15,0,15,15,15,15,15},
                                                                                                                   {15,15,0,15,15,0,15,15,15,15},
                                                                                                                   {15,15,15,0,15,0,15,15,15,15},
                                                                                                                   {15,15,15,15,0,15,0,15,15,15},
                                                                                                                   {0,15,15,15,15,0,0,15,15,15},
                                                                                                                   {0,0,15,15,15,15,0,0,0,15},
                                                                                                                   {15,15,0,0,15,15,15,0,15,0}};
__constant__ double LocalNoise2D_2[INNER_Z_END_2 - INNER_Z_START_2 + 1][INNER_X_END_2 - INNER_X_START_2 + 1];




#define INNER_X_START_3 4
#define INNER_Z_START_3 0

#define INNER_X_END_3 15
#define INNER_Z_END_3 10

__constant__ uint8_t DIRT_HEIGHT_2D_3[INNER_Z_END_3 - INNER_Z_START_3 + 1][INNER_X_END_3 - INNER_X_START_3 + 1] = {{1,1,15,15,15,15,15,15,15,15,0,15},
                                                                                                                   {15,15,15,15,15,15,15,15,15,15,0,15},
                                                                                                                   {15,15,15,15,15,15,15,15,15,15,15,0},
                                                                                                                   {15,15,15,0,15,15,15,15,15,15,15,0},
                                                                                                                   {15,15,15,1,15,15,15,15,15,15,15,15},
                                                                                                                   {15,15,15,0,15,15,15,15,15,15,15,0},
                                                                                                                   {15,15,15,15,15,15,15,15,15,15,15,15},
                                                                                                                   {15,15,0,15,15,15,15,15,15,15,15,15},
                                                                                                                   {15,15,1,15,15,15,15,15,15,15,15,15},
                                                                                                                   {15,15,15,1,15,15,15,15,15,15,15,15},
                                                                                                                   {15,15,15,0,15,15,15,15,15,15,15,15}};
__constant__ double LocalNoise2D_3[INNER_Z_END_3 - INNER_Z_START_3 + 1][INNER_X_END_3 - INNER_X_START_3 + 1];
/*
//Old test: matches 104703450999364
#define CHUNK_X 2
#define CHUNK_Z 11

#define INNER_X_START 2
#define INNER_Z_START 0

#define INNER_X_END 11
#define INNER_Z_END 0


__constant__ uint8_t DIRT_HEIGHT_2D[INNER_Z_END - INNER_Z_START + 1][INNER_X_END - INNER_X_START + 1] = {{0,15,0,1,0,15,15,15,15,1}};
__constant__ double LocalNoise2D[INNER_Z_END - INNER_Z_START + 1][INNER_X_END - INNER_X_START + 1];
*/



//The generation of the simplex layers and noise
namespace noise { //region Simplex layer gen
    /* End of constant for simplex noise*/
    
    struct Octave {
        double xo;
        double yo;
        double zo;
        uint8_t permutations[256];
    };

    __shared__ uint8_t permutations[256][BLOCK_SIZE];


    #define getValue(array, index) array[index][threadIdx.x]
    #define setValue(array, index, value) array[index][threadIdx.x] = value


    __device__ static inline void setupNoise(const uint8_t nbOctaves, Random *random, Octave resultArray[]) {
        for (int j = 0; j < nbOctaves; ++j) {
            __prefetch_local_l2(&resultArray[j]);
            resultArray[j].xo = next_double(random) * 256.0;
            resultArray[j].yo = next_double(random) * 256.0;
            resultArray[j].zo = next_double(random) * 256.0;
            
            #pragma unroll
            for(int w = 0; w<256; w++) {
                setValue(permutations, w, w);
            }
            for(int index = 0; index<256; index++) {
                uint32_t randomIndex = random_next_int(random, 256ull - index) + index;
                //if (randomIndex != index) {
                    // swap
                    uint8_t v1 = getValue(permutations,index);
                    //uint8_t v2 = getValue(permutations,randomIndex);
                    setValue(permutations,index, getValue(permutations,randomIndex));
                    setValue(permutations, randomIndex, v1);
                //}
            }
            #pragma unroll
            for(int c = 0; c<256;c++) {
                __prefetch_local_l1(&(resultArray[j].permutations[c+1]));
                resultArray[j].permutations[c] = getValue(permutations,c);
            }
            //resultArray[j].xo = xo;
            //resultArray[j].yo = yo;
            //resultArray[j].zo = zo;
        }
    }
    __device__ static inline void SkipNoiseGen(const uint8_t nbOctaves, Random* random) {
        for (int j = 0; j < nbOctaves; ++j) {
            lcg::advance<2*3>(*random);
            for(int index = 0; index<256; index++) {
                random_next_int(random, 256ull - index);
            }
        }
    }
    
    __device__ static inline double lerp(double x, double a, double b) {
        return a + x * (b - a);
    }

    __device__ static inline double grad(uint8_t hash, double x, double y, double z) {
        switch (hash & 0xFu) {
            case 0x0:
                return x + y;
            case 0x1:
                return -x + y;
            case 0x2:
                return x - y;
            case 0x3:
                return -x - y;
            case 0x4:
                return x + z;
            case 0x5:
                return -x + z;
            case 0x6:
                return x - z;
            case 0x7:
                return -x - z;
            case 0x8:
                return y + z;
            case 0x9:
                return -y + z;
            case 0xA:
                return y - z;
            case 0xB:
                return -y - z;
            case 0xC:
                return y + x;
            case 0xD:
                return -y + z;
            case 0xE:
                return y - x;
            case 0xF:
                return -y - z;
            default:
                return 0; // never happens
        }
    }


    __device__ static inline void generateNormalPermutations(double *buffer, double x, double y, double z, int sizeX, int sizeY, int sizeZ, double noiseFactorX, double noiseFactorY, double noiseFactorZ, double octaveSize, Random* random) {
        double xo = lcg::next_double(*random) * 256.0;
		double yo = lcg::next_double(*random) * 256.0;
		double zo = lcg::next_double(*random) * 256.0;
		//Setup the permutation fresh xD
		#pragma unroll
		for(int w = 0; w<256; w++) {
			setValue(permutations, w, w);
		}
		for(int index = 0; index<256; index++) {
			uint32_t randomIndex = lcg::dynamic_next_int(*random, 256ull - index) + index;
			//if (randomIndex != index) {
				// swap
				uint8_t v1 = getValue(permutations,index);
				uint8_t v2 = getValue(permutations,randomIndex);
				setValue(permutations,index, v2);
				setValue(permutations, randomIndex, v1);
			//}
		}
		
		double octaveWidth = 1.0 / octaveSize;
        int32_t i2 = -1;
        double x1 = 0.0;
        double x2 = 0.0;
        double xx1 = 0.0;
        double xx2 = 0.0;
        double t;
        double w;
        int columnIndex = 0;
        for (int X = 0; X < sizeX; X++) {
            double xCoord = (x + (double) X) * noiseFactorX + xo;
            auto clampedXcoord = (int32_t) xCoord;
            if (xCoord < (double) clampedXcoord) {
                clampedXcoord--;
            }
            auto xBottoms = (uint8_t) ((uint32_t) clampedXcoord & 0xffu);
            xCoord -= clampedXcoord;
            t = xCoord * 6 - 15;
            w = (xCoord * t + 10);
            double fadeX = xCoord * xCoord * xCoord * w;
            for (int Z = 0; Z < sizeZ; Z++) {
                double zCoord = zo;
                auto clampedZCoord = (int32_t) zCoord;
                if (zCoord < (double) clampedZCoord) {
                    clampedZCoord--;
                }
                auto zBottoms = (uint8_t) ((uint32_t) clampedZCoord & 0xffu);
                zCoord -= clampedZCoord;
                t = zCoord * 6 - 15;
                w = (zCoord * t + 10);
                double fadeZ = zCoord * zCoord * zCoord * w;
                for (int Y = 0; Y < sizeY; Y++) {
                    double yCoords = (y + (double) Y) * noiseFactorY + yo;
                    auto clampedYCoords = (int32_t) yCoords;
                    if (yCoords < (double) clampedYCoords) {
                        clampedYCoords--;
                    }
                    auto yBottoms = (uint8_t) ((uint32_t) clampedYCoords & 0xffu);
                    yCoords -= clampedYCoords;
                    t = yCoords * 6 - 15;
                    w = yCoords * t + 10;
                    double fadeY = yCoords * yCoords * yCoords * w;
                    // ZCoord

                    if (Y == 0 || yBottoms != i2) { // this is wrong on so many levels, same ybottoms doesnt mean x and z were the same...
						i2 = yBottoms;
						uint16_t k2 = getValue(permutations,(uint8_t)((uint16_t)(getValue(permutations,(uint8_t)(xBottoms& 0xffu)) + yBottoms)& 0xffu)) + zBottoms;
						uint16_t l2 = getValue(permutations,(uint8_t)((uint16_t)(getValue(permutations,(uint8_t)(xBottoms& 0xffu)) + yBottoms + 1u )& 0xffu)) + zBottoms;
						uint16_t k3 = getValue(permutations,(uint8_t)((uint16_t)(getValue(permutations,(uint8_t)((xBottoms + 1u)& 0xffu)) + yBottoms )& 0xffu)) + zBottoms;
						uint16_t l3 = getValue(permutations,(uint8_t)((uint16_t)(getValue(permutations,(uint8_t)((xBottoms + 1u)& 0xffu)) + yBottoms + 1u) & 0xffu)) + zBottoms;
						x1 = lerp(fadeX, grad(getValue(permutations,(uint8_t)(k2& 0xffu)), xCoord, yCoords, zCoord), grad(getValue(permutations,(uint8_t)(k3& 0xffu)), xCoord - 1.0, yCoords, zCoord));
						x2 = lerp(fadeX, grad(getValue(permutations,(uint8_t)(l2& 0xffu)), xCoord, yCoords - 1.0, zCoord), grad(getValue(permutations,(uint8_t)(l3& 0xffu)), xCoord - 1.0, yCoords - 1.0, zCoord));
						xx1 = lerp(fadeX, grad(getValue(permutations,(uint8_t)((k2+1u)& 0xffu)), xCoord, yCoords, zCoord - 1.0), grad(getValue(permutations,(uint8_t)((k3+1u)& 0xffu)), xCoord - 1.0, yCoords, zCoord - 1.0));
						xx2 = lerp(fadeX, grad(getValue(permutations,(uint8_t)((l2+1u)& 0xffu)), xCoord, yCoords - 1.0, zCoord - 1.0), grad(getValue(permutations,(uint8_t)((l3+1u)& 0xffu)), xCoord - 1.0, yCoords - 1.0, zCoord - 1.0));
					}

                    if (columnIndex%16 >= INNER_X_START && columnIndex%16 <= INNER_X_END &&
                        DIRT_HEIGHT_2D[columnIndex/16 - INNER_Z_START][columnIndex%16 - INNER_X_START] != 15){
                        double y1 = lerp(fadeY, x1, x2);
                        double y2 = lerp(fadeY, xx1, xx2);
                        (buffer)[columnIndex] = (buffer)[columnIndex] + lerp(fadeZ, y1, y2) * octaveWidth;
                    }

                    if (columnIndex == EARLY_RETURN) return;
                    
                    columnIndex++;

                }
            }
        }
    }


    __device__ static inline void generateNormalPermutations_2(double *buffer, double x, double y, double z, int sizeX, int sizeY, int sizeZ, double noiseFactorX, double noiseFactorY, double noiseFactorZ, double octaveSize, Random* random) {
        double xo = lcg::next_double(*random) * 256.0;
		double yo = lcg::next_double(*random) * 256.0;
		double zo = lcg::next_double(*random) * 256.0;
		//Setup the permutation fresh xD
		#pragma unroll
		for(int w = 0; w<256; w++) {
			setValue(permutations, w, w);
		}
		for(int index = 0; index<256; index++) {
			uint32_t randomIndex = lcg::dynamic_next_int(*random, 256ull - index) + index;
			//if (randomIndex != index) {
				// swap
				uint8_t v1 = getValue(permutations,index);
				uint8_t v2 = getValue(permutations,randomIndex);
				setValue(permutations,index, v2);
				setValue(permutations, randomIndex, v1);
			//}
		}
		double octaveWidth = 1.0 / octaveSize;
        int32_t i2 = -1;
        double x1 = 0.0;
        double x2 = 0.0;
        double xx1 = 0.0;
        double xx2 = 0.0;
        double t;
        double w;
        int columnIndex = 0;
        for (int X = 0; X < sizeX; X++) {
            double xCoord = (x + (double) X) * noiseFactorX + xo;
            auto clampedXcoord = (int32_t) xCoord;
            if (xCoord < (double) clampedXcoord) {
                clampedXcoord--;
            }
            auto xBottoms = (uint8_t) ((uint32_t) clampedXcoord & 0xffu);
            xCoord -= clampedXcoord;
            t = xCoord * 6 - 15;
            w = (xCoord * t + 10);
            double fadeX = xCoord * xCoord * xCoord * w;
            for (int Z = 0; Z < sizeZ; Z++) {
                double zCoord = zo;
                auto clampedZCoord = (int32_t) zCoord;
                if (zCoord < (double) clampedZCoord) {
                    clampedZCoord--;
                }
                auto zBottoms = (uint8_t) ((uint32_t) clampedZCoord & 0xffu);
                zCoord -= clampedZCoord;
                t = zCoord * 6 - 15;
                w = (zCoord * t + 10);
                double fadeZ = zCoord * zCoord * zCoord * w;
                for (int Y = 0; Y < sizeY; Y++) {
                    double yCoords = (y + (double) Y) * noiseFactorY + yo;
                    auto clampedYCoords = (int32_t) yCoords;
                    if (yCoords < (double) clampedYCoords) {
                        clampedYCoords--;
                    }
                    auto yBottoms = (uint8_t) ((uint32_t) clampedYCoords & 0xffu);
                    yCoords -= clampedYCoords;
                    t = yCoords * 6 - 15;
                    w = yCoords * t + 10;
                    double fadeY = yCoords * yCoords * yCoords * w;
                    // ZCoord

                    if (Y == 0 || yBottoms != i2) { // this is wrong on so many levels, same ybottoms doesnt mean x and z were the same...
						i2 = yBottoms;
						uint16_t k2 = getValue(permutations,(uint8_t)((uint16_t)(getValue(permutations,(uint8_t)(xBottoms& 0xffu)) + yBottoms)& 0xffu)) + zBottoms;
						uint16_t l2 = getValue(permutations,(uint8_t)((uint16_t)(getValue(permutations,(uint8_t)(xBottoms& 0xffu)) + yBottoms + 1u )& 0xffu)) + zBottoms;
						uint16_t k3 = getValue(permutations,(uint8_t)((uint16_t)(getValue(permutations,(uint8_t)((xBottoms + 1u)& 0xffu)) + yBottoms )& 0xffu)) + zBottoms;
						uint16_t l3 = getValue(permutations,(uint8_t)((uint16_t)(getValue(permutations,(uint8_t)((xBottoms + 1u)& 0xffu)) + yBottoms + 1u) & 0xffu)) + zBottoms;
						x1 = lerp(fadeX, grad(getValue(permutations,(uint8_t)(k2& 0xffu)), xCoord, yCoords, zCoord), grad(getValue(permutations,(uint8_t)(k3& 0xffu)), xCoord - 1.0, yCoords, zCoord));
						x2 = lerp(fadeX, grad(getValue(permutations,(uint8_t)(l2& 0xffu)), xCoord, yCoords - 1.0, zCoord), grad(getValue(permutations,(uint8_t)(l3& 0xffu)), xCoord - 1.0, yCoords - 1.0, zCoord));
						xx1 = lerp(fadeX, grad(getValue(permutations,(uint8_t)((k2+1u)& 0xffu)), xCoord, yCoords, zCoord - 1.0), grad(getValue(permutations,(uint8_t)((k3+1u)& 0xffu)), xCoord - 1.0, yCoords, zCoord - 1.0));
						xx2 = lerp(fadeX, grad(getValue(permutations,(uint8_t)((l2+1u)& 0xffu)), xCoord, yCoords - 1.0, zCoord - 1.0), grad(getValue(permutations,(uint8_t)((l3+1u)& 0xffu)), xCoord - 1.0, yCoords - 1.0, zCoord - 1.0));
					}

                    if (columnIndex%16 >= INNER_X_START_2 && columnIndex%16 <= INNER_X_END_2 &&
                        DIRT_HEIGHT_2D_2[columnIndex/16 - INNER_Z_START_2][columnIndex%16 - INNER_X_START_2] != 15){
                        double y1 = lerp(fadeY, x1, x2);
                        double y2 = lerp(fadeY, xx1, xx2);
                        (buffer)[columnIndex] = (buffer)[columnIndex] + lerp(fadeZ, y1, y2) * octaveWidth;
                    }
                    
                    columnIndex++;

                }
            }
        }
    }

    __device__ static inline void generateNormalPermutations_3(double *buffer, double x, double y, double z, int sizeX, int sizeY, int sizeZ, double noiseFactorX, double noiseFactorY, double noiseFactorZ, double octaveSize, Random* random) {
        double xo = lcg::next_double(*random) * 256.0;
		double yo = lcg::next_double(*random) * 256.0;
		double zo = lcg::next_double(*random) * 256.0;
		//Setup the permutation fresh xD
		#pragma unroll
		for(int w = 0; w<256; w++) {
			setValue(permutations, w, w);
		}
		for(int index = 0; index<256; index++) {
			uint32_t randomIndex = lcg::dynamic_next_int(*random, 256ull - index) + index;
			//if (randomIndex != index) {
				// swap
				uint8_t v1 = getValue(permutations,index);
				uint8_t v2 = getValue(permutations,randomIndex);
				setValue(permutations,index, v2);
				setValue(permutations, randomIndex, v1);
			//}
		}
		double octaveWidth = 1.0 / octaveSize;
        int32_t i2 = -1;
        double x1 = 0.0;
        double x2 = 0.0;
        double xx1 = 0.0;
        double xx2 = 0.0;
        double t;
        double w;
        int columnIndex = 0;
        for (int X = 0; X < sizeX; X++) {
            double xCoord = (x + (double) X) * noiseFactorX + xo;
            auto clampedXcoord = (int32_t) xCoord;
            if (xCoord < (double) clampedXcoord) {
                clampedXcoord--;
            }
            auto xBottoms = (uint8_t) ((uint32_t) clampedXcoord & 0xffu);
            xCoord -= clampedXcoord;
            t = xCoord * 6 - 15;
            w = (xCoord * t + 10);
            double fadeX = xCoord * xCoord * xCoord * w;
            for (int Z = 0; Z < sizeZ; Z++) {
                double zCoord = zo;
                auto clampedZCoord = (int32_t) zCoord;
                if (zCoord < (double) clampedZCoord) {
                    clampedZCoord--;
                }
                auto zBottoms = (uint8_t) ((uint32_t) clampedZCoord & 0xffu);
                zCoord -= clampedZCoord;
                t = zCoord * 6 - 15;
                w = (zCoord * t + 10);
                double fadeZ = zCoord * zCoord * zCoord * w;
                for (int Y = 0; Y < sizeY; Y++) {
                    double yCoords = (y + (double) Y) * noiseFactorY + yo;
                    auto clampedYCoords = (int32_t) yCoords;
                    if (yCoords < (double) clampedYCoords) {
                        clampedYCoords--;
                    }
                    auto yBottoms = (uint8_t) ((uint32_t) clampedYCoords & 0xffu);
                    yCoords -= clampedYCoords;
                    t = yCoords * 6 - 15;
                    w = yCoords * t + 10;
                    double fadeY = yCoords * yCoords * yCoords * w;
                    // ZCoord

                    if (Y == 0 || yBottoms != i2) { // this is wrong on so many levels, same ybottoms doesnt mean x and z were the same...
						i2 = yBottoms;
						uint16_t k2 = getValue(permutations,(uint8_t)((uint16_t)(getValue(permutations,(uint8_t)(xBottoms& 0xffu)) + yBottoms)& 0xffu)) + zBottoms;
						uint16_t l2 = getValue(permutations,(uint8_t)((uint16_t)(getValue(permutations,(uint8_t)(xBottoms& 0xffu)) + yBottoms + 1u )& 0xffu)) + zBottoms;
						uint16_t k3 = getValue(permutations,(uint8_t)((uint16_t)(getValue(permutations,(uint8_t)((xBottoms + 1u)& 0xffu)) + yBottoms )& 0xffu)) + zBottoms;
						uint16_t l3 = getValue(permutations,(uint8_t)((uint16_t)(getValue(permutations,(uint8_t)((xBottoms + 1u)& 0xffu)) + yBottoms + 1u) & 0xffu)) + zBottoms;
						x1 = lerp(fadeX, grad(getValue(permutations,(uint8_t)(k2& 0xffu)), xCoord, yCoords, zCoord), grad(getValue(permutations,(uint8_t)(k3& 0xffu)), xCoord - 1.0, yCoords, zCoord));
						x2 = lerp(fadeX, grad(getValue(permutations,(uint8_t)(l2& 0xffu)), xCoord, yCoords - 1.0, zCoord), grad(getValue(permutations,(uint8_t)(l3& 0xffu)), xCoord - 1.0, yCoords - 1.0, zCoord));
						xx1 = lerp(fadeX, grad(getValue(permutations,(uint8_t)((k2+1u)& 0xffu)), xCoord, yCoords, zCoord - 1.0), grad(getValue(permutations,(uint8_t)((k3+1u)& 0xffu)), xCoord - 1.0, yCoords, zCoord - 1.0));
						xx2 = lerp(fadeX, grad(getValue(permutations,(uint8_t)((l2+1u)& 0xffu)), xCoord, yCoords - 1.0, zCoord - 1.0), grad(getValue(permutations,(uint8_t)((l3+1u)& 0xffu)), xCoord - 1.0, yCoords - 1.0, zCoord - 1.0));
					}

                    if (columnIndex%16 >= INNER_X_START_3 && columnIndex%16 <= INNER_X_END_3 &&
                        DIRT_HEIGHT_2D_3[columnIndex/16 - INNER_Z_START_3][columnIndex%16 - INNER_X_START_3] != 15){
                        double y1 = lerp(fadeY, x1, x2);
                        double y2 = lerp(fadeY, xx1, xx2);
                        (buffer)[columnIndex] = (buffer)[columnIndex] + lerp(fadeZ, y1, y2) * octaveWidth;
                    }
                    
                    columnIndex++;

                }
            }
        }
    }


    __device__ static inline void generateNoise(double *buffer, double chunkX, double chunkY, double chunkZ, int sizeX, int sizeY, int sizeZ, double offsetX, double offsetY, double offsetZ, Random random, int nbOctaves) {
        //memset(buffer, 0, sizeof(double) * sizeX * sizeZ * sizeY);
        double octavesFactor = 1.0;
        for (int octave = 0; octave < nbOctaves; octave++) {
            generateNormalPermutations(buffer, chunkX, chunkY, chunkZ, sizeX, sizeY, sizeZ, offsetX * octavesFactor, offsetY * octavesFactor, offsetZ * octavesFactor, octavesFactor, &random);
            octavesFactor /= 2.0;
        }
    }

    __device__ static inline void generateNoise_2(double *buffer, double chunkX, double chunkY, double chunkZ, int sizeX, int sizeY, int sizeZ, double offsetX, double offsetY, double offsetZ, Random random, int nbOctaves) {
        //memset(buffer, 0, sizeof(double) * sizeX * sizeZ * sizeY);
        double octavesFactor = 1.0;
        for (int octave = 0; octave < nbOctaves; octave++) {
            generateNormalPermutations_2(buffer, chunkX, chunkY, chunkZ, sizeX, sizeY, sizeZ, offsetX * octavesFactor, offsetY * octavesFactor, offsetZ * octavesFactor, octavesFactor, &random);
            octavesFactor /= 2.0;
        }
    }

    __device__ static inline void generateNoise_3(double *buffer, double chunkX, double chunkY, double chunkZ, int sizeX, int sizeY, int sizeZ, double offsetX, double offsetY, double offsetZ, Random random, int nbOctaves) {
        //memset(buffer, 0, sizeof(double) * sizeX * sizeZ * sizeY);
        double octavesFactor = 1.0;
        for (int octave = 0; octave < nbOctaves; octave++) {
            generateNormalPermutations_3(buffer, chunkX, chunkY, chunkZ, sizeX, sizeY, sizeZ, offsetX * octavesFactor, offsetY * octavesFactor, offsetZ * octavesFactor, octavesFactor, &random);
            octavesFactor /= 2.0;
        }
    }
}
using namespace noise;


__device__ static inline bool match(uint64_t seed) {
    seed = get_random(seed);
    //SkipNoiseGen(16+16+8, &seed);
    lcg::advance<10480>(seed);//VERY VERY DODGY
    
    
    double heightField[EARLY_RETURN+1];
    #pragma unroll
    for(uint16_t i = 0; i<EARLY_RETURN+1;i++)
        heightField[i] = 0;
    
    const double noiseFactor = 0.03125;
    generateNoise(heightField, (double) (CHUNK_X <<4), (double) (CHUNK_Z<<4), 0.0, 16, 16, 1, noiseFactor, noiseFactor, 1.0, seed, 4);

    for(uint8_t z = 0; z < INNER_Z_END - INNER_Z_START + 1; z++) {
        for(uint8_t x = 0; x < INNER_X_END - INNER_X_START + 1; x++) {
            if (DIRT_HEIGHT_2D[z][x] != 15) {
                uint8_t dirty = heightField[INNER_X_START + x + (INNER_Z_START + z) * 16] + LocalNoise2D[z][x] * 0.2 > 0.0 ? 0 : 1;
                if (dirty!=(int8_t)DIRT_HEIGHT_2D[z][x]) 
                    return false;
            }
        }
    }
    return true;
}

__device__ static inline bool match2(uint64_t seed) {
    seed = get_random(seed);
    //SkipNoiseGen(16+16+8, &seed);
    lcg::advance<10480>(seed);//VERY VERY DODGY
    
    double heightField[256];
    #pragma unroll
    for(uint16_t i = 0; i<256;i++)
        heightField[i] = 0;
    
    const double noiseFactor = 0.03125;
    generateNoise_2(heightField, (double) (CHUNK_X_2 <<4), (double) (CHUNK_Z_2<<4), 0.0, 16, 16, 1, noiseFactor, noiseFactor, 1.0, seed, 4);

    for(uint8_t z = 0; z < INNER_Z_END_2 - INNER_Z_START_2 + 1; z++) {
        for(uint8_t x = 0; x < INNER_X_END_2 - INNER_X_START_2 + 1; x++) {
            if (DIRT_HEIGHT_2D_2[z][x] != 15) {
                uint8_t dirty = heightField[INNER_X_START_2 + x + (INNER_Z_START_2 + z) * 16] + LocalNoise2D_2[z][x] * 0.2 > 0.0 ? 0 : 1;
                if (dirty!=(int8_t)DIRT_HEIGHT_2D_2[z][x]) 
                    return false;
            }
        }
    }
    return true;
}

__device__ static inline bool match3(uint64_t seed) {
    seed = get_random(seed);
    //SkipNoiseGen(16+16+8, &seed);
    lcg::advance<10480>(seed);//VERY VERY DODGY
    
    
    double heightField[256];
    #pragma unroll
    for(uint16_t i = 0; i<256;i++)
        heightField[i] = 0;
    
    const double noiseFactor = 0.03125;
    generateNoise_3(heightField, (double) (CHUNK_X_3 <<4), (double) (CHUNK_Z_3<<4), 0.0, 16, 16, 1, noiseFactor, noiseFactor, 1.0, seed, 4);

    for(uint8_t z = 0; z < INNER_Z_END_3 - INNER_Z_START_3 + 1; z++) {
        for(uint8_t x = 0; x < INNER_X_END_3 - INNER_X_START_3 + 1; x++) {
            if (DIRT_HEIGHT_2D_3[z][x] != 15) {
                uint8_t dirty = heightField[INNER_X_START_3 + x + (INNER_Z_START_3 + z) * 16] + LocalNoise2D_3[z][x] * 0.2 > 0.0 ? 0 : 1;
                if (dirty!=(int8_t)DIRT_HEIGHT_2D_3[z][x]) 
                    return false;
            }
        }
    }
    return true;
}


__global__ __launch_bounds__(BLOCK_SIZE,2) static void tempCheck(uint64_t offset, uint64_t* buffer, uint32_t* counter) {
    uint64_t seed = blockIdx.x * blockDim.x + threadIdx.x + offset;

    if (match(seed)) {
        buffer[atomicAdd(counter,1)] = seed;
    }
}

__global__ __launch_bounds__(BLOCK_SIZE,2) static void tempCheck2(uint32_t count, uint64_t* buffer) {
    uint64_t seedIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (seedIndex>=count)
        return;
    if (!match2(buffer[seedIndex])) {
        buffer[seedIndex] = 0;
    }
}

__global__ __launch_bounds__(BLOCK_SIZE,2) static void tempCheck3(uint32_t count, uint64_t* buffer) {
    uint64_t seedIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (seedIndex>=count)
        return;
    uint64_t seed = buffer[seedIndex];
    if (seed==0)
        return;
    if (!match3(seed)) {
        buffer[seedIndex] = 0;
    }
}






std::ifstream inSeeds;
std::ofstream outSeeds;

uint64_t* buffer;
uint32_t* counter;

double getNextDoubleForLocNoise(int x, int z);
void setup(int gpu_device) {
    hipSetDevice(gpu_device);
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    
    double locNoise2D[INNER_Z_END - INNER_Z_START + 1][INNER_X_END - INNER_X_START + 1];
    for(uint8_t z = 0; z < INNER_Z_END - INNER_Z_START + 1; z++) {
        for (uint8_t x = 0; x < INNER_X_END - INNER_X_START + 1; x++) {
            locNoise2D[z][x] = getNextDoubleForLocNoise((CHUNK_X<<4) + INNER_X_START + x, (CHUNK_Z<<4) + INNER_Z_START + z);
        }
    }

    GPU_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(LocalNoise2D), &locNoise2D, sizeof(locNoise2D)));
    GPU_ASSERT(hipPeekAtLastError());
    
    double locNoise2D_2[INNER_Z_END_2 - INNER_Z_START_2 + 1][INNER_X_END_2 - INNER_X_START_2 + 1];
    for(uint8_t z = 0; z < INNER_Z_END_2 - INNER_Z_START_2 + 1; z++) {
        for (uint8_t x = 0; x < INNER_X_END_2 - INNER_X_START_2 + 1; x++) {
            locNoise2D_2[z][x] = getNextDoubleForLocNoise((CHUNK_X_2<<4) + INNER_X_START_2 + x, (CHUNK_Z_2<<4) + INNER_Z_START_2 + z);
        }
    }

    GPU_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(LocalNoise2D_2), &locNoise2D_2, sizeof(locNoise2D_2)));
    GPU_ASSERT(hipPeekAtLastError());
    
    double locNoise2D_3[INNER_Z_END_3 - INNER_Z_START_3 + 1][INNER_X_END_3 - INNER_X_START_3 + 1];
    for(uint8_t z = 0; z < INNER_Z_END_3 - INNER_Z_START_3 + 1; z++) {
        for (uint8_t x = 0; x < INNER_X_END_3 - INNER_X_START_3 + 1; x++) {
            locNoise2D_3[z][x] = getNextDoubleForLocNoise((CHUNK_X_3<<4) + INNER_X_START_3 + x, (CHUNK_Z_3<<4) + INNER_Z_START_3 + z);
        }
    }

    GPU_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(LocalNoise2D_3), &locNoise2D_3, sizeof(locNoise2D_3)));
    GPU_ASSERT(hipPeekAtLastError());
}

time_t elapsed_chkpoint = 0;
struct checkpoint_vars {
    unsigned long long offset;
    time_t elapsed_chkpoint;
};

int main(int argc, char *argv[]) {
    int gpu_device = 0;
    uint64_t START;
    uint64_t offsetStart = 0;
    uint64_t COUNT;
    int x = 116;
    int chunkxCPU = 6;
    int chunkxCPU2 = 6;
    int chunkxCPU3 = 5;
	#ifdef BOINC
    BOINC_OPTIONS options;
    boinc_options_defaults(options);
	options.normal_thread_priority = true;
    boinc_init_options(&options);
    #endif
	for (int i = 1; i < argc; i += 2) {
		const char *param = argv[i];
		if (strcmp(param, "-d") == 0 || strcmp(param, "--device") == 0) {
			gpu_device = atoi(argv[i + 1]);
		} else if (strcmp(param, "-s") == 0 || strcmp(param, "--start") == 0) {
			sscanf(argv[i + 1], "%llu", &START);
		} else if (strcmp(param, "-c") == 0 || strcmp(param, "--count") == 0) {
			sscanf(argv[i + 1], "%llu", &COUNT);
        } else if (strcmp(param, "-x") == 0){
            sscanf(argv[i + 1], "%i", &x);
        } 
        else {
			fprintf(stderr,"Unknown parameter: %s\n", param);
		}
    }
    x = (x>>4) - 7;
    chunkxCPU += x;
    chunkxCPU2 += x;
    chunkxCPU3 += x;
    GPU_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(CHUNK_X), &chunkxCPU, sizeof(CHUNK_X)));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(CHUNK_X_2), &chunkxCPU2, sizeof(CHUNK_X_2)));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(CHUNK_X_3), &chunkxCPU3, sizeof(CHUNK_X_3)));
    GPU_ASSERT(hipPeekAtLastError());
    FILE *checkpoint_data = boinc_fopen("packpoint.txt", "rb");

    if(!checkpoint_data){
        fprintf(stderr, "No checkpoint to load\n");

    }
    else{
        #ifdef BOINC
            boinc_begin_critical_section();
        #endif

        struct checkpoint_vars data_store;
        fread(&data_store, sizeof(data_store), 1, checkpoint_data);
        offsetStart = data_store.offset;
        elapsed_chkpoint = data_store.elapsed_chkpoint;
        fprintf(stderr, "Checkpoint loaded, task time %d s, seed pos: %llu\n", elapsed_chkpoint, START);
        fclose(checkpoint_data);
        #ifdef BOINC
            boinc_end_critical_section();
        #endif
    }
	#ifdef BOINC
	APP_INIT_DATA aid;
	boinc_get_init_data(aid);
	
	if (aid.gpu_device_num >= 0) {
		gpu_device = aid.gpu_device_num;
		fprintf(stderr,"boinc gpu %i gpuindex: %i \n", aid.gpu_device_num, gpu_device);
		} else {
		fprintf(stderr,"stndalone gpuindex %i \n", gpu_device);
	}
	#endif
    setup(gpu_device);
    uint64_t seedCount = COUNT;
    std::cout << "Processing " << seedCount << " seeds" << std::endl;

    outSeeds.open("seedsout");
    GPU_ASSERT(hipMallocManaged(&buffer, sizeof(*buffer) * SEEDS_PER_CALL));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipMallocManaged(&counter, sizeof(*counter)));
    GPU_ASSERT(hipPeekAtLastError());
    time_t start_time = time(NULL);
    int outCount = 0;

    int checkpointTemp = 0;
    for(uint64_t offset =offsetStart;offset<seedCount;offset+=SEEDS_PER_CALL) {
        // Normal filtering
        time_t elapsed = time(NULL) - start_time;
        double frac = (double) offset / (double)(seedCount);
        #ifdef BOINC
            boinc_fraction_done(frac);
        #endif
        *counter = 0;
        tempCheck<<<1ULL<<WORK_SIZE_BITS,BLOCK_SIZE>>>(START + offset, buffer,counter);
        GPU_ASSERT(hipPeekAtLastError());
        GPU_ASSERT(hipDeviceSynchronize());  
        
        tempCheck2<<<((*counter)/BLOCK_SIZE)+1,BLOCK_SIZE>>>(*counter, buffer);
        GPU_ASSERT(hipPeekAtLastError());
        GPU_ASSERT(hipDeviceSynchronize());
        
        tempCheck3<<<((*counter)/BLOCK_SIZE)+1,BLOCK_SIZE>>>(*counter, buffer);
        GPU_ASSERT(hipPeekAtLastError());
        GPU_ASSERT(hipDeviceSynchronize());
        
        for(int i=0;i<*counter;i++) {
            if (buffer[i]!=0) {
                uint64_t seed = buffer[i];
                std::cout << "3rd level seed found: " << seed << std::endl;
                outSeeds << seed << std::endl;
                outCount++;
            }
        }

        if(checkpointTemp >= 180000000 || boinc_time_to_checkpoint()){
            #ifdef BOINC
		        boinc_begin_critical_section(); // Boinc should not interrupt this
            #endif
            // Checkpointing section below
			boinc_delete_file("packpoint.txt"); // Don't touch, same func as normal fdel
            FILE *checkpoint_data = boinc_fopen("packpoint.txt", "wb");
			struct checkpoint_vars data_store;
			data_store.offset = offset;
            data_store.elapsed_chkpoint = elapsed_chkpoint + elapsed;
            fwrite(&data_store, sizeof(data_store), 1, checkpoint_data);
            fclose(checkpoint_data);
            checkpointTemp = 0;
            #ifdef BOINC
            boinc_end_critical_section();
            boinc_checkpoint_completed(); // Checkpointing completed
            #endif
        }
        checkpointTemp += SEEDS_PER_CALL;
        std::cout << "Seeds left:" << (((int64_t)seedCount-offset)-SEEDS_PER_CALL) << std::endl;  
    }

    std::cout << "Done processing" << std::endl;    
    #ifdef BOINC
	    boinc_begin_critical_section();
	#endif
    time_t elapsed = time(NULL) - start_time;
    double done = (double)COUNT / 1000000.0;
    double speed = done / (double) elapsed;
    fprintf(stderr, "\nSpeed: %.2lfm/s\n", speed );
    fprintf(stderr, "Done\n");
    fprintf(stderr, "Processed: %llu seeds in %.2lfs seconds\n", COUNT, (double) elapsed_chkpoint + (double) elapsed );
    fprintf(stderr, "Have %llu output seeds.\n", outCount);
    fflush(stderr);
    outSeeds.close();
    boinc_delete_file("packpoint.txt");
    #ifdef BOINC
        boinc_end_critical_section();
    #endif
    boinc_finish(0);
}

double getNextDoubleForLocNoise(int x, int z) {
    Random rand = get_random((((int64_t)x) >> 4) * 341873128712LL + (((int64_t)z) >> 4) * 132897987541LL);
    for (int dx = 0; dx < 16; dx++) {
      for (int dz = 0; dz < 16; dz++) {
        if (dx == (x & 15) && dz == (z & 15)) {
          //advance2(&rand);
          //advance2(&rand);
          return next_double(&rand);
        }
        advance2(&rand);
        advance2(&rand);
        advance2(&rand);
        for(int k1 = 127; k1 >= 0; k1--) {
          random_next_int_nonpow(&rand,5);
        }
        //for (int i = 0; i < 67; i++) {
        //  advance2(&rand);
        //}
      }
    }
    exit(-99);
}
